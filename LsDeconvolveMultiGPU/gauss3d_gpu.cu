#include "hip/hip_runtime.h"
/*
    gauss3d_gpu.cu

    High-performance, in-place 3D Gaussian filtering for MATLAB gpuArray inputs.

    ----------------------------------------------------------------------------
    Author:       Keivan Moradi (with assistance from ChatGPT v4.1, 2025)
    License:      GPL v3
    ----------------------------------------------------------------------------

    Overview:
    ---------
    This MEX function implements a fast, memory-efficient, block-wise **3D Gaussian filter**
    for single-precision (`single`) 3D gpuArray data in MATLAB, using CUDA for GPU acceleration.
    It is **API-compatible** with MATLAB's `imgaussfilt3`, but is highly optimized
    for batch processing and large volumes, and designed to be integrated into GPU deconvolution pipelines.

    **IMPORTANT: This version performs filtering IN-PLACE. The input gpuArray is
    overwritten and returned as output. The original input data will be destroyed.**

    Key Features:
    -------------
      - **In-place destructive filtering:** Minimizes VRAM usage by modifying the input array directly.
        Only a single workspace buffer (same size as the array) is allocated in addition to the input.
      - **Heavy CUDA optimization:** Performs separable convolution along all 3 axes using constant-memory kernels, and launches tuned CUDA kernels for maximum performance.
      - **Workspace control:** Accepts user-provided block padding and kernel size to allow batch-wise processing (important for large volumes or integration in multi-step GPU workflows).
      - **OOM-resilient:** Attempts memory allocation with automatic retries and helpful warnings when out-of-memory occurs.
      - **MATLAB gpuArray interface:** Input and output are both MATLAB `gpuArray(single)` objects, fully compatible with native MATLAB workflows.
      - **Flexible sigma and kernel size:** Accepts scalar or vector `sigma` and kernel size for anisotropic filtering.
      - **Open source, GPL v3**.

    Differences from MATLAB's imgaussfilt3:
    -----------------------------------------
      1. **Much faster** on large data: Algorithm is hand-optimized for GPU with memory reuse and minimal transfers.
      2. **Destructive in-place operation:** The input gpuArray is modified and returned. This avoids allocating a second full-size array and reduces VRAM requirements by up to 33%.
      3. **External workspace control:** Padding/batching is managed outside the function, making it suitable for tiled processing during deconvolution or large-scale pipelines.
      4. **Separable convolution:** Uses 1D convolutions in 3 passes, exploiting constant memory for kernel coefficients.
      5. **Direct gpuArray support:** Does not require conversion or intermediate CPU copies.

    Usage Example (in MATLAB):
    --------------------------
        x = gpuArray(single(randn(128,128,64)));
        y = gauss3d_gpu(x, 2.0);               % x is destroyed/overwritten; y is the filtered result
        y = gauss3d_gpu(x, [2 1 4], [9 5 15]); % Anisotropic sigma & kernel size

    Notes:
    ------
      - Input must be a 3D `gpuArray` of single precision.
      - The function is **destructive**: the input will be overwritten.
      - Designed for block-wise and pipelined use, e.g., in deconvolution, denoising, or pre-processing.
      - All main computation is performed on the GPU with minimal synchronization overhead.

    Acknowledgments:
    ----------------
      - Original algorithm and MEX/CUDA optimizations by Keivan Moradi.
      - ChatGPT (OpenAI GPT-4.1, 2025) provided structural and code review assistance.

*/


#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cmath>
#include <cstdio>
#include <thread>   // For std::this_thread::sleep_for
#include <chrono>   // For std::chrono::milliseconds
#include <hipfft/hipfft.h>
#include <vector>

#define CUDA_CHECK(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        mexWarnMsgIdAndTxt("gauss3d_gpu:cuda", "CUDA error %s:%d: %s", __FILE__, __LINE__, hipGetErrorString(err)); \
    } \
} while(0)

#define MAX_KERNEL_SIZE 51
__constant__ float const_kernel_f[MAX_KERNEL_SIZE];

// Gaussian kernel creation (host)
void make_gaussian_kernel(float sigma, int ksize, float* kernel) {
    int r = ksize / 2;
    double sum = 0.0;
    for (int i = -r; i <= r; ++i) {
        kernel[i + r] = static_cast<float>(std::exp(-0.5 * (i * i) / (sigma * sigma)));
        sum += kernel[i + r];
    }
    for (int i = 0; i < ksize; ++i)
        kernel[i] = static_cast<float>(kernel[i] / sum);
}

// CUDA 1D convolution kernel for float (with restrict)
__global__ void gauss1d_kernel_const_float(
    const float* __restrict__ src, float* __restrict__ dst,
    size_t nx, size_t ny, size_t nz,
    int klen, int axis)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    size_t nline, linelen;
    if (axis == 0) { linelen = nx; nline = ny * nz; }
    else if (axis == 1) { linelen = ny; nline = nx * nz; }
    else { linelen = nz; nline = nx * ny; }
    if (tid >= nline * linelen) return;

    size_t line = tid / linelen;
    size_t pos = tid % linelen;

    size_t x, y, z;
    if (axis == 0) {
        y = line % ny;
        z = line / ny;
        x = pos;
    } else if (axis == 1) {
        x = line % nx;
        z = line / nx;
        y = pos;
    } else {
        x = line % nx;
        y = line / nx;
        z = pos;
    }

    size_t idx = x + y * nx + z * nx * ny;
    int r = klen / 2;
    float acc = 0.0f;
    for (int s = 0; s < klen; ++s) {
        int offset = s - r;
        int xi = static_cast<int>(x);
        int yi = static_cast<int>(y);
        int zi = static_cast<int>(z);
        if (axis == 0) xi = min(max(static_cast<int>(x) + offset, 0), static_cast<int>(nx) - 1);
        if (axis == 1) yi = min(max(static_cast<int>(y) + offset, 0), static_cast<int>(ny) - 1);
        if (axis == 2) zi = min(max(static_cast<int>(z) + offset, 0), static_cast<int>(nz) - 1);
        size_t src_idx = xi + yi * nx + zi * nx * ny;
        acc += src[src_idx] * const_kernel_f[s];
    }
    dst[idx] = acc;
}

// Host orchestration for float
void gauss3d_separable_float(
    float* input,
    float* buffer,
    size_t nx, size_t ny, size_t nz,
    const float sigma[3], const int ksize[3],
    bool* error_flag)
{
    int max_klen = std::max({ksize[0], ksize[1], ksize[2]});
    if (max_klen > MAX_KERNEL_SIZE) {
        mexWarnMsgIdAndTxt("gauss3d_gpu:ksize", "Kernel size exceeds MAX_KERNEL_SIZE (%d)", MAX_KERNEL_SIZE);
        if (error_flag) *error_flag = true;
        return;
    }
    float* h_kernel = new float[max_klen];
    float* src = input;
    float* dst = buffer;
    bool local_error = false;

    // --- Use hipOccupancyMaxPotentialBlockSize for kernel launch tuning ---
    int minGrid, blockSize;
    hipOccupancyMaxPotentialBlockSize(&minGrid, &blockSize, gauss1d_kernel_const_float, 0, 0);

    for (int axis = 0; axis < 3; ++axis) {
        make_gaussian_kernel(sigma[axis], ksize[axis], h_kernel);
        hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(const_kernel_f), h_kernel, ksize[axis] * sizeof(float), 0, hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            mexWarnMsgIdAndTxt("gauss3d_gpu:cuda", "CUDA memcpyToSymbol error: %s", hipGetErrorString(err));
            local_error = true;
            break;
        }
        size_t linelen = (axis == 0) ? nx : (axis == 1) ? ny : nz;
        size_t nline   = (axis == 0) ? ny * nz : (axis == 1) ? nx * nz : nx * ny;
        size_t total = linelen * nline;
        int grid = static_cast<int>((total + blockSize - 1) / blockSize);

        gauss1d_kernel_const_float<<<grid, blockSize, 0>>>(
            src, dst, nx, ny, nz, ksize[axis], axis);

        err = hipGetLastError();
        if (err != hipSuccess) {
            mexWarnMsgIdAndTxt("gauss3d_gpu:cuda", "CUDA kernel launch error: %s", hipGetErrorString(err));
            local_error = true;
            break;
        }
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            mexWarnMsgIdAndTxt("gauss3d_gpu:cuda", "CUDA device synchronize error: %s", hipGetErrorString(err));
            local_error = true;
            break;
        }
        std::swap(src, dst);
    }

    if (!local_error && src != input) {
        hipError_t err = hipMemcpy(input, src, nx * ny * nz * sizeof(float), hipMemcpyDeviceToDevice);
        if (err != hipSuccess) {
            mexWarnMsgIdAndTxt("gauss3d_gpu:cuda", "CUDA memcpy result error: %s", hipGetErrorString(err));
            local_error = true;
        }
    }

    delete[] h_kernel;
    if (error_flag) *error_flag = local_error;
}

// A small helper to check CUFFT calls:
#define CUFFT_CHECK(call) do {                            \
    hipfftResult err = call;                               \
    if (err != HIPFFT_SUCCESS) {                           \
        mexWarnMsgIdAndTxt(                               \
            "gauss3d_gpu:fft",                            \
            "CUFFT error %d at %s:%d",                    \
            (int)err, __FILE__, __LINE__                  \
        );                                                \
        goto cleanup;                                     \
    }                                                     \
} while(0)

// --- CUDA kernel helpers ---
__global__ void mult_freq_domain(hipfftComplex* a, const hipfftComplex* b, int n) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n) {
        hipfftComplex x = a[i], y = b[i];
        a[i].x = x.x*y.x - x.y*y.y;
        a[i].y = x.x*y.y + x.y*y.x;
    }
}
__global__ void scale_array(float* a, int n, float scale) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n) a[i] *= scale;
}

// Column-major linear index: row + col*rows + slice*rows*cols
static inline __host__ __device__
size_t cmIdx(int row,int col,int z,int rows,int cols)
{
    return static_cast<size_t>(row)
         + static_cast<size_t>(col)*rows
         + static_cast<size_t>(z)  *rows*cols;
}

static inline size_t linIdx(int x,int y,int z,int nx,int ny)
{ return static_cast<size_t>(x) + static_cast<size_t>(y)*nx
         + static_cast<size_t>(z)*nx*ny; }

// -----------------------------------------------------------------------------
// Build a full-volume Gaussian kernel:
//
// • Truncate at user-supplied ksize (half-widths rx,ry,rz)
// • Normalise so sum == 1
// • Circularly shift by −⌊N/2⌋ in each dimension so the centre lands at (0,0,0)
// • Memory layout matches MATLAB (column-major).
// -----------------------------------------------------------------------------
void make_gaussian_kernel_fft(float*       kernel,
                              int          nRows,    // MATLAB dim-1  (Y)
                              int          nCols,    // MATLAB dim-2  (X)
                              int          nZ,
                              const float  sigma[3], // σx,σy,σz per imgaussfilt3
                              const int    ksize[3]) // user-requested odd sizes
{
    const int rCentre = nRows / 2;
    const int cCentre = nCols / 2;
    const int zCentre = nZ    / 2;

    const int ry = ksize[1] / 2;   // NOTE: σ[1] is along rows (Y)
    const int rx = ksize[0] / 2;   // σ[0] along columns (X)
    const int rz = ksize[2] / 2;

    double sum = 0.0;

    // -------- fill (row,col,z) ------------------------------------------------
    for (int z = 0; z < nZ; ++z)
    {
        const int dz = z - zCentre;
        const float dz_n = (sigma[2] > 0.f) ? dz / sigma[2] : 0.f;

        for (int col = 0; col < nCols; ++col)
        {
            const int dc = col - cCentre;
            const float dc_n = (sigma[0] > 0.f) ? dc / sigma[0] : 0.f;

            for (int row = 0; row < nRows; ++row)
            {
                const int dr = row - rCentre;
                float val = 0.f;

                if (std::abs(dc) <= rx &&
                    std::abs(dr) <= ry &&
                    std::abs(dz) <= rz)
                {
                    const float dr_n = (sigma[1] > 0.f) ? dr / sigma[1] : 0.f;
                    val = expf(-0.5f * (dc_n*dc_n + dr_n*dr_n + dz_n*dz_n));
                }

                kernel[ cmIdx(row,col,z,nRows,nCols) ] = val;
                sum += val;
            }
        }
    }

    // -------- normalise -------------------------------------------------------
    if (sum > 0.0)
    {
        const float invSum = static_cast<float>(1.0 / sum);
        const size_t total = static_cast<size_t>(nRows)*nCols*nZ;
        for (size_t i = 0; i < total; ++i) kernel[i] *= invSum;
    }

    // -------- circular shift by −⌊N/2⌋ --------------------------------------
    const int sRow = nRows / 2;   // MATLAB: floor(N/2)
    const int sCol = nCols / 2;
    const int sZ   = nZ    / 2;

    std::vector<float> tmp(kernel, kernel + static_cast<size_t>(nRows)*nCols*nZ);

    for (int z = 0; z < nZ; ++z)
    {
        const int srcZ = (z + sZ) % nZ;
        for (int col = 0; col < nCols; ++col)
        {
            const int srcC = (col + sCol) % nCols;
            for (int row = 0; row < nRows; ++row)
            {
                const int srcR = (row + sRow) % nRows;
                kernel[ cmIdx(row,col,z,nRows,nCols) ] =
                    tmp   [ cmIdx(srcR,srcC,srcZ,nRows,nCols) ];
            }
        }
    }
}

// -----------------------------------------------------------------------------
// FFT-based Gaussian filtering (periodic boundary conditions).
// Out-of-place, safe for use with MATLAB gpuArray buffers.
// -----------------------------------------------------------------------------
void gauss3d_fft_float(float* d_input, int nx, int ny, int nz,
                       const float sigma[3], const int ksize[3],
                       bool* error_flag)
{
    // ---- Shapes (column-major order): [nx, ny, nz] in MATLAB ===
    // cuFFT expects [nz, ny, nx] (depth, height, width)
    const size_t N      = (size_t)nx * ny * nz;              // Real volume
    const int    NZfreq = nz / 2 + 1;                        // C2R/R2C output: freq dim is Z (slowest)
    const size_t Nfreq  = (size_t)NZfreq * ny * nx;          // cuFFT complex freq size

    if (error_flag) *error_flag = true;

    float*        d_kernel    = nullptr;
    hipfftComplex* d_freq_in   = nullptr;
    hipfftComplex* d_freq_ker  = nullptr;
    hipfftHandle   planR2C     = 0;
    hipfftHandle   planC2R     = 0;
    float*        d_result    = nullptr;

    try {
        // ---- Device allocations ----
        CUDA_CHECK(hipMalloc(&d_kernel,  sizeof(float) * N));
        CUDA_CHECK(hipMalloc(&d_freq_in, sizeof(hipfftComplex) * Nfreq));
        CUDA_CHECK(hipMalloc(&d_freq_ker,sizeof(hipfftComplex) * Nfreq));
        CUDA_CHECK(hipMalloc(&d_result,  sizeof(float) * N));

        // ---- Build & upload kernel ----
        std::vector<float> h_kernel(N, 0.f);
        make_gaussian_kernel_fft(h_kernel.data(), nx, ny, nz, sigma, ksize);
        CUDA_CHECK(hipMemcpy(d_kernel, h_kernel.data(),
                              sizeof(float) * N, hipMemcpyHostToDevice));

        // ---- FFT plans: dims = [nz, ny, nx] ----
        CUFFT_CHECK(hipfftPlan3d(&planR2C, nz, ny, nx, HIPFFT_R2C));
        CUFFT_CHECK(hipfftPlan3d(&planC2R, nz, ny, nx, HIPFFT_C2R));

        // ---- Forward FFTs ----
        CUFFT_CHECK(hipfftExecR2C(planR2C, d_input,  d_freq_in));
        CUFFT_CHECK(hipfftExecR2C(planR2C, d_kernel, d_freq_ker));

        // ---- Point-wise multiply in frequency domain ----
        constexpr int THREADS = 256;
        int blocks = static_cast<int>((Nfreq + THREADS - 1) / THREADS);
        mult_freq_domain<<<blocks, THREADS>>>(d_freq_in, d_freq_ker, static_cast<int>(Nfreq));
        CUDA_CHECK(hipGetLastError());

        // ---- Inverse FFT and normalize ----
        CUFFT_CHECK(hipfftExecC2R(planC2R, d_freq_in, d_result));
        blocks = static_cast<int>((N + THREADS - 1) / THREADS);
        scale_array<<<blocks, THREADS>>>(d_result, static_cast<int>(N), 1.f / float(N));
        CUDA_CHECK(hipGetLastError());

        // ---- Copy result back to input buffer (for in-place output) ----
        CUDA_CHECK(hipMemcpy(d_input, d_result, sizeof(float) * N, hipMemcpyDeviceToDevice));

        if (error_flag) *error_flag = false;

    } catch (...) {
        mexWarnMsgIdAndTxt("gauss3d_gpu:fft", "Exception during FFT filtering");
        if (error_flag) *error_flag = true;
    }

    // ---- Cleanup ----
cleanup:
    if (planR2C)   hipfftDestroy(planR2C);
    if (planC2R)   hipfftDestroy(planC2R);
    if (d_kernel)  hipFree(d_kernel);
    if (d_freq_in) hipFree(d_freq_in);
    if (d_freq_ker)hipFree(d_freq_ker);
    if (d_result)  hipFree(d_result);
}


// ================
// MEX entry point
// ================
extern "C" void mexFunction(int nlhs, mxArray* plhs[], int nrhs, const mxArray* prhs[]) {
    mxInitGPU();
    float* buffer = nullptr;
    bool error_flag = false;
    mxGPUArray* img_gpu = nullptr;

    try {
        if (nrhs < 2)
            mexErrMsgIdAndTxt("gauss3d_gpu:", "Usage: gauss3d_gpu(x, sigma [, kernel_size])");

        img_gpu = (mxGPUArray*)mxGPUCreateFromMxArray(prhs[0]);
        const mwSize* sz = mxGPUGetDimensions(img_gpu);
        int nd = mxGPUGetNumberOfDimensions(img_gpu);
        if (nd != 3)
            mexErrMsgIdAndTxt("gauss3d_gpu:", "Input must be 3D.");

        size_t nx = (size_t)sz[0], ny = (size_t)sz[1], nz = (size_t)sz[2];
        size_t N = nx * ny * nz;
        mxClassID cls = mxGPUGetClassID(img_gpu);
        void* ptr = mxGPUGetData(img_gpu);

        if (cls != mxSINGLE_CLASS)
            mexErrMsgIdAndTxt("gauss3d_gpu:", "Input must be single-precision gpuArray");

        double sigma_double[3];
        if (mxIsScalar(prhs[1])) {
            double v = mxGetScalar(prhs[1]);
            sigma_double[0] = sigma_double[1] = sigma_double[2] = v;
        } else if (mxGetNumberOfElements(prhs[1]) == 3) {
            double* s = mxGetPr(prhs[1]);
            for (int i = 0; i < 3; ++i) sigma_double[i] = s[i];
        } else {
            mexErrMsgIdAndTxt("gauss3d_gpu:", "sigma must be scalar or 3-vector");
        }

        int ksize[3];
        if (nrhs >= 3 && !mxIsLogicalScalar(prhs[2])) {
            if (mxIsEmpty(prhs[2])) {
                for (int i = 0; i < 3; ++i)
                    ksize[i] = 2 * (int)ceil(3.0 * sigma_double[i]) + 1;
            } else if (mxIsScalar(prhs[2])) {
                int k = (int)mxGetScalar(prhs[2]);
                ksize[0] = ksize[1] = ksize[2] = k;
            } else if (mxGetNumberOfElements(prhs[2]) == 3) {
                double* ks = mxGetPr(prhs[2]);
                for (int i = 0; i < 3; ++i) ksize[i] = (int)ks[i];
            } else {
                mexErrMsgIdAndTxt("gauss3d_gpu:", "kernel_size must be scalar or 3-vector");
            }
        } else {
            for (int i = 0; i < 3; ++i)
                ksize[i] = 2 * (int)ceil(3.0 * sigma_double[i]) + 1;
        }

        // --------- Allocate workspace buffer only ---------
        int max_retries = 2;
        int retries = 0;
        hipError_t alloc_err;
        while (retries < max_retries) {
            alloc_err = hipMalloc(&buffer, N * sizeof(float));
            if (alloc_err == hipSuccess && buffer != nullptr)
                break;
            size_t free_bytes = 0, total_bytes = 0;
            hipMemGetInfo(&free_bytes, &total_bytes);
            mexWarnMsgIdAndTxt("gauss3d_gpu:cuda",
                "gauss3d_gpu: CUDA OOM: Tried to allocate %.2f MB (Free: %.2f MB). Attempt %d/%d.",
                N * sizeof(float) / 1024.0 / 1024.0,
                free_bytes / 1024.0 / 1024.0,
                retries + 1, max_retries);
            hipDeviceSynchronize();
            std::this_thread::sleep_for(std::chrono::milliseconds(1000));
            retries++;
        }
        if (alloc_err != hipSuccess || !buffer) {
            mexErrMsgIdAndTxt("gauss3d_gpu:cuda",
                "gauss3d_gpu: CUDA OOM: Could not allocate workspace buffer (%.2f MB) after %d attempts.",
                N * sizeof(float) / 1024.0 / 1024.0, max_retries);
        }

        float sigma[3] = { (float)sigma_double[0], (float)sigma_double[1], (float)sigma_double[2] };

        // --------- In-place filtering or FFT! ---------
        float* data_ptr = static_cast<float*>(ptr);
        
        bool use_fft = false;
        int fft_size_thresh = 256; // cube root of 16 million; tune for your GPU
        
        if (nrhs >= 4 && mxIsChar(prhs[3])) {
            char mode_buf[16];
            mxGetString(prhs[3], mode_buf, sizeof(mode_buf));
            if (strcmp(mode_buf, "fft") == 0) use_fft = true;
            else if (strcmp(mode_buf, "direct") == 0) use_fft = false;
            // else auto
        } else {
            // Auto-switch
            use_fft = (nx >= fft_size_thresh || ny >= fft_size_thresh || nz >= fft_size_thresh
                        || std::max({ksize[0], ksize[1], ksize[2]}) > 31);
        }
        
        if (use_fft) {
            gauss3d_fft_float(      data_ptr, nx, ny, nz, sigma, ksize, &error_flag);
        } else {
            gauss3d_separable_float(data_ptr, buffer, nx, ny, nz, sigma, ksize, &error_flag);
        }

        CUDA_CHECK(hipDeviceSynchronize());

        // Return the (modified) input gpuArray as output
        plhs[0] = mxGPUCreateMxArrayOnGPU(img_gpu);

    } catch (...) {
        mexPrintf("gauss3d_gpu: Unknown error! Possible OOM or kernel failure.\n");
        error_flag = true;
    }

    // ----------- CLEANUP (always reached) --------------
    if (buffer)
        hipFree(buffer);
    // Only destroy img_gpu if NOT returned to MATLAB
    // This is robust: if plhs[0] has been set, MATLAB owns it.
    if (img_gpu && (nlhs == 0 || (plhs[0] != mxGPUCreateMxArrayOnGPU(img_gpu))))
        mxGPUDestroyGPUArray(img_gpu);
}
